#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cctype>
#include <unordered_map>
#include <unordered_set>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;
using namespace std::chrono;

// Host function to normalize text
string normalize(const string &input) {
    string result = input;
    transform(result.begin(), result.end(), result.begin(), ::tolower);
    return result;
}

// Host function to build LPS array
vector<int> buildLPS(const string &pattern) {
    int m = pattern.length();
    vector<int> lps(m, 0);
    int len = 0, i = 1;
    
    while (i < m) {
        if (pattern[i] == pattern[len]) {
            lps[i] = ++len;
            i++;
        } else {
            if (len != 0) {
                len = lps[len - 1];
            } else {
                lps[i] = 0;
                i++;
            }
        }
    }
    return lps;
}

// CUDA kernel to perform KMP search in parallel (one thread per pattern)
__global__ void KMPSearchKernel(char* text, int textLen, char** patterns, int* patternLens, 
                               int* lps_arrays, int* lps_offsets, char* results, int numPatterns) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < numPatterns) {
        char* pattern = patterns[tid];
        int patternLen = patternLens[tid];
        int* lps = &lps_arrays[lps_offsets[tid]];
        
        int i = 0; // index for text
        int j = 0; // index for pattern
        
        // KMP algorithm
        while (i < textLen) {
            if (pattern[j] == text[i]) {
                i++;
                j++;
            }
            
            if (j == patternLen) {
                // Pattern found
                results[tid] = 1;
                break;
            } else if (i < textLen && pattern[j] != text[i]) {
                if (j != 0)
                    j = lps[j - 1];
                else
                    i++;
            }
        }
    }
}

// Function to allocate memory and prepare data for CUDA
bool KMPSearchCUDA(const string &text, const vector<string> &patterns, vector<char> &results) {
    int numPatterns = patterns.size();
    results.resize(numPatterns, 0);
    
    // Allocate host memory
    vector<char*> h_patterns(numPatterns);
    vector<int> h_patternLens(numPatterns);
    vector<int> h_lps_offsets(numPatterns, 0);
    vector<int> h_lps_arrays;
    
    // Prepare LPS arrays and pattern data
    int total_lps_size = 0;
    for (int i = 0; i < numPatterns; i++) {
        h_patternLens[i] = patterns[i].length();
        
        // Build LPS array for this pattern
        vector<int> lps = buildLPS(patterns[i]);
        
        // Store offset
        h_lps_offsets[i] = total_lps_size;
        
        // Append to the flattened array
        h_lps_arrays.insert(h_lps_arrays.end(), lps.begin(), lps.end());
        total_lps_size += lps.size();
    }
    
    // Allocate device memory
    char* d_text;
    char** d_patterns;
    int* d_patternLens;
    int* d_lps_arrays;
    int* d_lps_offsets;
    char* d_results;
    
    hipMalloc(&d_text, text.length() * sizeof(char));
    hipMalloc(&d_patterns, numPatterns * sizeof(char*));
    hipMalloc(&d_patternLens, numPatterns * sizeof(int));
    hipMalloc(&d_lps_arrays, total_lps_size * sizeof(int));
    hipMalloc(&d_lps_offsets, numPatterns * sizeof(int));
    hipMalloc(&d_results, numPatterns * sizeof(char));
    
    // Allocate pattern strings on device
    vector<char*> d_pattern_ptrs(numPatterns);
    for (int i = 0; i < numPatterns; i++) {
        hipMalloc(&d_pattern_ptrs[i], patterns[i].length() * sizeof(char));
        hipMemcpy(d_pattern_ptrs[i], patterns[i].c_str(), patterns[i].length() * sizeof(char), hipMemcpyHostToDevice);
    }
    
    // Copy data to device
    hipMemcpy(d_text, text.c_str(), text.length() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_patterns, d_pattern_ptrs.data(), numPatterns * sizeof(char*), hipMemcpyHostToDevice);
    hipMemcpy(d_patternLens, h_patternLens.data(), numPatterns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lps_arrays, h_lps_arrays.data(), total_lps_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_lps_offsets, h_lps_offsets.data(), numPatterns * sizeof(int), hipMemcpyHostToDevice);
    
    // Initialize results to false (0)
    hipMemset(d_results, 0, numPatterns * sizeof(char));
    
    // Launch kernel
    int blockSize = 256;
    int numBlocks = (numPatterns + blockSize - 1) / blockSize;
    
    KMPSearchKernel<<<numBlocks, blockSize>>>(d_text, text.length(), d_patterns, d_patternLens, 
                                             d_lps_arrays, d_lps_offsets, d_results, numPatterns);
    
    // Copy results back
    hipMemcpy(results.data(), d_results, numPatterns * sizeof(char), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_text);
    hipFree(d_patterns);
    hipFree(d_patternLens);
    hipFree(d_lps_arrays);
    hipFree(d_lps_offsets);
    hipFree(d_results);
    
    for (int i = 0; i < numPatterns; i++) {
        hipFree(d_pattern_ptrs[i]);
    }
    
    
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        cerr << "CUDA error: " << hipGetErrorString(error) << endl;
        return false;
    }
    
    return true;
}

// Risk classification function
string classifyRisk(int riskScore) {
    if (riskScore <= 30)
        return "low";
    else if (riskScore <= 60)
        return "medium";
    else if (riskScore <= 80)
        return "high";
    return "critical";
}

int main() {
    // Define keyword weights (same as in original code)
    unordered_map<string, int> keywordWeights = {
        {"; drop table ", 100},
        {"xp_cmdshell", 100},
        {"outfile '/", 100},
        {"load_file('/", 100},
        {"delete", 15},
        {"insert", 15},
        {"truncate", 15},
        {"update", 15},
        {"alter", 15},
        {"or 1=1", 10},
        {"--", 10},
        {"#", 10},
        {"/*", 10},
        {"*/", 10},
        {"sleep(", 10},
        {"benchmark(", 10},
        {"union", 5},
        {"select", 5},
        {"exists(", 10},
        {"* from users", 10},
        {"select username password from users", 10},
        {"union select", 10},
        {"version(", 20},
        {"current_user", 20},
        {"database(", 25},
        {"information_schema.schemata", 25}
    };

    // Create a vector of patterns from the dictionary
    vector<string> sqli_patterns;
    for (const auto &p : keywordWeights) {
        sqli_patterns.push_back(p.first);
    }

    // Open the CSV File
    ifstream infile("sqli_dataset_Mid_New.csv");
    if (!infile.is_open()) {
        cerr << "Error: Could not open the CSV file." << endl;
        return 1;
    }

    vector<string> queries;
    vector<string> expected_risks;
    
    string line;
    while (getline(infile, line)) {
        if (line.empty()) continue;
        
        stringstream ss(line);
        string query, expectedRisk, expectedScore;
        
        if (!getline(ss, query, ',')) continue;
        if (!getline(ss, expectedRisk, ',')) continue;
        getline(ss, expectedScore, ',');
        
        queries.push_back(query);
        expected_risks.push_back(expectedRisk);
    }
    infile.close();
    
    int totalQueries = queries.size();
    int correctCount = 0;
    
    // Normalize patterns
    vector<string> normalized_patterns;
    for (const string &pattern : sqli_patterns) {
        normalized_patterns.push_back(normalize(pattern));
    }
    
    // Start timing
    auto start = high_resolution_clock::now();
    
    // Process each query
    for (int q = 0; q < totalQueries; q++) {
        string query = queries[q];
        string expectedRisk = expected_risks[q];
        
        // Normalize query
        string normQuery = normalize(query);
        
        // Vector to store results of parallel KMP search
        vector<char> searchResults;
        
        // Perform CUDA KMP search
        KMPSearchCUDA(normQuery, normalized_patterns, searchResults);
        
        // Calculate risk score based on results
        int riskScore = 0;
        unordered_set<string> foundPatterns;
        
        for (int i = 0; i < normalized_patterns.size(); i++) {
            if (searchResults[i] == 1) {
                string pattern = sqli_patterns[i];
                if (foundPatterns.find(pattern) == foundPatterns.end()) {
                    foundPatterns.insert(pattern);
                    riskScore += keywordWeights[pattern];
                }
            }
        }
        
        // Classify risk
        string computedRisk = classifyRisk(riskScore);
        bool match = (computedRisk == expectedRisk);
        
        if (match) {
            correctCount++;
        }
        
        // Print results for first few queries only
        if (q < 5) {
            cout << "Query: " << query << endl;
            cout << "Score: " << riskScore << endl;
            cout << "Expected Risk: " << expectedRisk << " | Computed Risk: " << computedRisk << endl;
            cout << (match ? "Match" : "Mismatch") << "\n--------------------------" << endl;
        }
    }
    
    // End timing
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(end - start).count();
    
    cout << "\nTotal Queries Processed: " << totalQueries << endl;
    cout << "Matching Classifications: " << correctCount << endl;
    double accuracy = (totalQueries > 0) ? (100.0 * correctCount / totalQueries) : 0.0;
    cout << "Accuracy: " << accuracy << "%" << endl;
    cout << "Execution Time: " << duration << " ms" << endl;
    
    return 0;
}